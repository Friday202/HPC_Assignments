#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "lenia.h"
#include "orbium.h"
#include "gifenc.h"

// #include <omp.h>
// #include <mpi.h>
// #include <hip/hip_runtime.h>
// #include <hip/hip_runtime.h>

// Uncomment to generate gif animation
#define GENERATE_GIF

// For prettier indexing syntax
#define w(r, c) (w[(r) * w_cols + (c)])
#define input(r, c) (input[((r) % rows) * cols + ((c) % cols)])

// Function to calculate Gaussian
inline double gauss(double x, double mu, double sigma)
{
    return exp(-0.5 * pow((x - mu) / sigma, 2));
}

// Function for growth criteria
double growth_lenia(double u)
{
    double mu = 0.15;
    double sigma = 0.015;
    return -1 + 2 * gauss(u, mu, sigma); // Baseline -1, peak +1
}

// Function to generate convolution kernel
double *generate_kernel(double *K, const unsigned int size)
{
    // Construct ring convolution filter
    double mu = 0.5;
    double sigma = 0.15;
    int r = size / 2;
    double sum = 0;
    if (K != NULL)
    {
        for (int y = -r; y < r; y++)
        {
            for (int x = -r; x < r; x++)
            {
                double distance = sqrt((1 + x) * (1 + x) + (1 + y) * (1 + y)) / r;
                K[(y + r) * size + x + r] = gauss(distance, mu, sigma);
                if (distance > 1)
                {
                    K[(y + r) * size + x + r] = 0; // Cut at d=1
                }
                sum += K[(y + r) * size + x + r];
            }
        }
        // Normalize
        for (unsigned int y = 0; y < size; y++)
        {
            for (unsigned int x = 0; x < size; x++)
            {
                K[y * size + x] /= sum;
            }
        }
    }
    return K;
}

// Function to perform convolution on input using kernel w
inline double *convolve2d(double *result, const double *input, const double *w, const unsigned int rows, const unsigned int cols, const unsigned int w_rows, const unsigned int w_cols)
{
    if (result != NULL && input != NULL && w != NULL)
    {
        for (unsigned int i = 0; i < rows; i++)
        {
            for (unsigned int j = 0; j < cols; j++)
            {
                double sum = 0;
                for (int ki = w_rows - 1, kri = 0; ki >= 0; ki--, kri++)
                {
                    for (int kj = w_cols - 1, kcj = 0; kj >= 0; kj--, kcj++)
                    {
                        sum += w(ki, kj) * input((i - w_rows / 2 + rows + kri), (j - w_cols / 2 + cols + kcj));
                    }
                }
                result[i * cols + j] = sum;
            }
        }
    }
    return result;
}

// Function to evolve Lenia
double *evolve_lenia(const unsigned int rows, const unsigned int cols, const unsigned int steps, const double dt, const unsigned int kernel_size, const struct orbium_coo *orbiums, const unsigned int num_orbiums)
{

#ifdef GENERATE_GIF
    ge_GIF *gif = ge_new_gif(
        "lenia_sequential.gif",     /* file name */
        cols, rows,      /* canvas size */
        inferno_pallete, /*pallete*/
        8,               /* palette depth == log2(# of colors) */
        -1,              /* no transparency */
        0                /* infinite loop */
    );
#endif

    // Allocate memory
    double *w = (double *)calloc(kernel_size * kernel_size, sizeof(double));
    double *world = (double *)calloc(rows * cols, sizeof(double));
    double *tmp = (double *)calloc(rows * cols, sizeof(double));

    // Generate convolution kernel
    w=generate_kernel(w,kernel_size);

    // Place orbiums
    for (unsigned int o = 0; o < num_orbiums; o++)
    {
        world = place_orbium(world, rows, cols, orbiums[o].row, orbiums[o].col, orbiums[o].angle);
    }

    // Lenia Simulation
    for (unsigned int step = 0; step < steps; step++)
    {
        // Convolution
        tmp = convolve2d(tmp, world, w, rows, cols, kernel_size, kernel_size);
        
        // Evolution
        for (unsigned int i = 0; i < rows; i++)
        {
            for (unsigned int j = 0; j < cols; j++)
            {
                world[i * rows + j] += dt * growth_lenia(tmp[i * rows + j]);
                world[i * rows + j] = fmin(1, fmax(0, world[i * rows + j])); // Clip between 0 and 1
#ifdef GENERATE_GIF
                gif->frame[i * rows + j] = world[i * rows + j] * 255;
#endif
            }
        }
#ifdef GENERATE_GIF
        ge_add_frame(gif, 5);
#endif
    }
#ifdef GENERATE_GIF
    ge_close_gif(gif);
#endif
    return world;
}
