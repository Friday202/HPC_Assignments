#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <stdlib.h>

#include <iostream>
#include <string>
#include <filesystem>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>   
#include "hip/hip_runtime_api.h"

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image.h"
#include "stb_image_write.h"

#define COLOR_CHANNELS 0
#define DEBUG 0
#define CUDA_DEBUG 0
//#define PIXEL_VALUES (1 << sizeof(unsigned char) * 8)
#define PIXEL_VALUES 256

/* CPU code for checking purposes */
void calculate_new_pixel_intensities_cpu(int* histogram_red, int* histogram_green, int* histogram_blue, int width, int height)
{
    int min_red = 0;
    int min_green = 0;
    int min_blue = 0;
    
    // Find minimum value in each histogram
    for (int i = 0; i < PIXEL_VALUES; ++i)
    {
		if (min_red == 0 && histogram_red[i] != 0)
		{
			min_red = histogram_red[i];			
		}
        if (min_green == 0 && histogram_green[i] != 0)
        {
            min_green = histogram_green[i];           
        }
        if (min_blue == 0 && histogram_blue[i] != 0)
        {
            min_blue = histogram_blue[i];            
        }
    }
    
    for (int i = 0; i < PIXEL_VALUES; ++i)
	{
        histogram_red[i] = ((float)(histogram_red[i] - min_red) / (width * height - min_red)) * (PIXEL_VALUES - 1);
        histogram_green[i] = ((float)(histogram_green[i] - min_green) / (width * height - min_green)) * (PIXEL_VALUES - 1);
        histogram_blue[i] = ((float)(histogram_blue[i] - min_blue) / (width * height - min_blue)) * (PIXEL_VALUES - 1);		
	}    
}

void calculate_histogram_cpu(const unsigned char* image, int* histogram_red, int* histogram_green, int* histogram_blue, const int width, const int height, const int cpp)
{
    for (int i = 0; i < width * height * cpp; i+= cpp)
    {
		histogram_red[image[i]]++;
		histogram_green[image[i + 1]]++;
		histogram_blue[image[i + 2]]++;
	}		
}

void calculate_cumulative_histogram_cpu(int* histogram_red, int* histogram_green, int* histogram_blue)
{	
	for (int i = 1; i < PIXEL_VALUES; i++)
	{
        histogram_red[i] += histogram_red[i - 1];
		histogram_green[i] += histogram_green[i - 1];
		histogram_blue[i] += histogram_blue[i - 1];		
	}
}
/* ~ CPU code for checking purposes */

__global__ void calculate_new_pixel_intensities_kernel(int* histogram_red, int* histogram_green, int* histogram_blue, const int width, const int height)
{
    // LOG
    // if (blockIdx.x == 0 && threadIdx.x == 0)
	// {
    //     printf("Executing kernel: Calculate new pixel intensities\n");                  
	// }

    __shared__ int min_red;
    __shared__ int min_green;
    __shared__ int min_blue;

    __syncthreads();  

    // Each block computes its own local minimum values
    if (threadIdx.x < 3) {
        if (threadIdx.x == 0) {
            for (int i = 0; i < PIXEL_VALUES; i++) {
                if (histogram_red[i] != 0) {
                    min_red = histogram_red[i];
                    break;
                }
            }
        }
        else if (threadIdx.x == 1) {
            for (int i = 0; i < PIXEL_VALUES; i++) {
                if (histogram_green[i] != 0) {
                    min_green = histogram_green[i];
                    break;
                }
            }
        }
        else if (threadIdx.x == 2) {
            for (int i = 0; i < PIXEL_VALUES; i++) {
                if (histogram_blue[i] != 0) {
                    min_blue = histogram_blue[i];
                    break;
                }
            }
        }
    }

    __syncthreads();       

    // int total_pixels = width * height;
    int global_thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    // TODO: I don't really understand the loop indexes 
    for (int i = global_thread_id; i < PIXEL_VALUES; i += blockDim.x * gridDim.x)
    {
        histogram_red[i] = ((float)(histogram_red[i] - min_red) / (width * height - min_red)) * (PIXEL_VALUES - 1);
        histogram_green[i] = ((float)(histogram_green[i] - min_green) / (width * height - min_green)) * (PIXEL_VALUES - 1);
        histogram_blue[i] = ((float)(histogram_blue[i] - min_blue) / (width * height - min_blue)) * (PIXEL_VALUES - 1);
    }    
}

__global__ void map_new_pixel_intensities_kernel(const unsigned char* imageIn, unsigned char* imageOut, const int* histogram_red, const int* histogram_green, const int* histogram_blue, const int width, const int height, const int cpp)
{
    // LOG
	// if (blockIdx.x == 0 && threadIdx.x == 0)
	// {
	// 	printf("Executing kernel: Map new pixel intensities\n");
	// }

	int global_thread_id = blockIdx.x * blockDim.x + threadIdx.x;
	for (int i = global_thread_id; i < width * height * cpp; i += blockDim.x * gridDim.x)
	{
		if (i % cpp == 0)
		{
			imageOut[i] = histogram_red[imageIn[i]];
		}
		else if (i % cpp == 1)
		{
			imageOut[i] = histogram_green[imageIn[i]];
		}
		else if (i % cpp == 2)
		{
			imageOut[i] = histogram_blue[imageIn[i]];
		}
	}	
}

__global__ void calculate_cumulative_sum_kernel(int* histogram_red, int* histogram_green, int* histogram_blue) //Algorithm 2 Work-efficient parallel scan algorithm.
{   

    //copy to shared memory (DOES NOT IMPROVE PERFORMANCE, INFACT IT MAKES IT WORSE!)
    // __shared__ int shared_histogram_red[PIXEL_VALUES];
    // __shared__ int shared_histogram_green[PIXEL_VALUES];
    // __shared__ int shared_histogram_blue[PIXEL_VALUES];
    // shared_histogram_red[threadIdx.x] = histogram_red[threadIdx.x];
    // shared_histogram_green[threadIdx.x] = histogram_green[threadIdx.x];
    // shared_histogram_blue[threadIdx.x] = histogram_blue[threadIdx.x];
    // __syncthreads();

    int log_pixel_val = 8;

    for (int k = 0; k < log_pixel_val; k++)
    {   
        int drugi = (threadIdx.x << (k+1)) + ((1 << (k + 1)) - 1);
        if (drugi < PIXEL_VALUES)
        {
            int prvi = (threadIdx.x << (k+1)) + (1 << k) - 1;
            // printf("Thread %d (k=%d) (limit=%f): %d[%d] + %d[%d]\n", threadIdx.x, k, pow(2, log_pixel_val-1-k), prvi, histogram_red[prvi], drugi, histogram_red[drugi]);
            if (blockIdx.x == 0) {
                histogram_red[drugi] = histogram_red[prvi] + histogram_red[drugi];
            } else if (blockIdx.x == 1) {
                histogram_green[drugi] = histogram_green[prvi] + histogram_green[drugi];
            } else if (blockIdx.x == 2) {
                histogram_blue[drugi] = histogram_blue[prvi] + histogram_blue[drugi];
            }
            
        }
        __syncthreads();
    }
    for (int k = log_pixel_val; k > 0; k--)
    {

        int drugi = (threadIdx.x << k) + (1 << k) + (1 << (k-1)) - 1;
        if (drugi < PIXEL_VALUES)
        {
            int prvi = (threadIdx.x << k) + (1 << k) - 1;
            // printf("Thread %d (k=-%d) (limit=%f): %d[%d] + %d[%d]\n", threadIdx.x, k, pow(2, log_pixel_val-1-k), drugi, histogram_red[drugi], prvi, histogram_red[prvi]);
            if (blockIdx.x == 0) {
                histogram_red[drugi] = histogram_red[prvi] + histogram_red[drugi];
            } else if (blockIdx.x == 1) {
                histogram_green[drugi] = histogram_green[prvi] + histogram_green[drugi];
            } else if (blockIdx.x == 2) {
                histogram_blue[drugi] = histogram_blue[prvi] + histogram_blue[drugi];
            }
        }
        __syncthreads();
    }

    //copy back to global memory
    // histogram_red[threadIdx.x] = shared_histogram_red[threadIdx.x];
    // histogram_green[threadIdx.x] = shared_histogram_green[threadIdx.x];
    // histogram_blue[threadIdx.x] = shared_histogram_blue[threadIdx.x];
}

__global__ void calculate_cumulative_sum_simple_kernel(int* histogram_red, int* histogram_green, int* histogram_blue)
{    
    // LOG
    // if (blockIdx.x == 0 && threadIdx.x == 0)
    // {
    //     printf("Executing kernel: Calculate cumulative sum\n");
    // }

    // Simple solution - only first three threads calculate the cumulative sums
    if (threadIdx.x == 0 && blockIdx.x == 0)
    {        
        for (int i = 1; i < PIXEL_VALUES; i++)
        {
            histogram_red[i] += histogram_red[i - 1];
        }    
    }
    else if (threadIdx.x == 1 && blockIdx.x == 0)
    {
        for (int i = 1; i < PIXEL_VALUES; i++)
        {
			histogram_green[i] += histogram_green[i - 1];
        }
    }
    else if (threadIdx.x == 2 && blockIdx.x == 0)
    {
        for (int i = 1; i < PIXEL_VALUES; i++)
        {
            histogram_blue[i] += histogram_blue[i - 1];
        }
    }
}

__global__ void calculate_histogram_kernel(const unsigned char* imageData, int* histogram_red, int* histogram_green, int* histogram_blue, const int width, const int height, const int cpp)
{ 
    // LOG
    // if (blockIdx.x == 0 && threadIdx.x == 0)
	// {
	// 	printf("Executing kernel: Calculate histogram\n");
	// }

    // Shared memory for partial histograms within block
    __shared__ int partial_histogram_red[PIXEL_VALUES];
    __shared__ int partial_histogram_green[PIXEL_VALUES];
    __shared__ int partial_histogram_blue[PIXEL_VALUES];

    // For each thread in block, parallel set partial histogram to 0
    for (int i = threadIdx.x; i < PIXEL_VALUES; i += blockDim.x) // blockDim.x always goes between 32 - 1024 
	{
        partial_histogram_red[i] = 0;
        partial_histogram_green[i] = 0;
        partial_histogram_blue[i] = 0;
	}    

    __syncthreads();

#if CUDA_DEBUG
    if (threadIdx.x == 0)
    {
        int sum = 0;
        for (int i = 0; i < PIXEL_VALUES; i++)
        {
            sum += partial_histogram[i];
        }
        if (sum != 0)
		{
			printf("Block %d: Partial histogram is not zeroed out\n", blockIdx.x);
            printf("Block %d: Partial histogram sum is %d\n", blockIdx.x, sum);
		}    
        if (blockIdx.x == 0)
        {
            for (int i = 0; i < PIXEL_VALUES; i++)
            {
				printf("First block histogram value %d: %d\n", i, partial_histogram[i]);
			}
		}
	}
    __syncthreads();
#endif    

    // TODO: Improve partial histogram so it doesn't jump around in memory
    // Calculate partial histogram, this probably causes a lot of cache misses
    int global_thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = global_thread_id; i < width * height * cpp; i += blockDim.x * gridDim.x)
	{
		if (i % cpp == 0)
		{
			atomicAdd(&partial_histogram_red[imageData[i]], 1);
		}
		else if (i % cpp == 1)
		{
			atomicAdd(&partial_histogram_green[imageData[i]], 1);
		}
		else if (i % cpp == 2)
		{
			atomicAdd(&partial_histogram_blue[imageData[i]], 1);
		}
	}   

    __syncthreads();

#if CUDA_DEBUG    
    if (threadIdx.x == 0)
    {
		int sum = 0;
        for (int i = 0; i < PIXEL_VALUES; i++)
        {
			sum += partial_histogram[i];
		}
		printf("Block %d: Partial histogram sum is %d\n", blockIdx.x, sum);
    }
    __syncthreads();
#endif  

    // Add aprtial histograms to global histogram
    // TODO: Possibly add reduction ? 
    for (int i = threadIdx.x; i < PIXEL_VALUES; i += blockDim.x) // blockDim.x always goes between 32 - 1024 
	{
		atomicAdd(&histogram_red[i], partial_histogram_red[i]);
		atomicAdd(&histogram_green[i], partial_histogram_green[i]);
		atomicAdd(&histogram_blue[i], partial_histogram_blue[i]);
	}
    
}

// example input arguments:
// C:\Users\Name\OneDrive\Namizje\FRI\HPC\CudaRuntime1\img\720x480.png C:\Users\Name\OneDrive\Namizje\FRI\HPC\CudaRuntime1\img\out.png
// h_ means host, d_ means device
int main(int argc, char* argv[])
{  
    if (argc < 3)
    {
        printf("USAGE: input_image output_image histBlockSize newPixelBlockSize mapPixelBlockSize\n");
        exit(EXIT_FAILURE);
    }

    char szImage_in_name[255];
    char szImage_out_name[255];

    snprintf(szImage_in_name, 255, "%s", argv[1]);
    snprintf(szImage_out_name, 255, "%s", argv[2]);
    
    // int histBlockSize = atoi(argv[3]);
    // int newPixelBlockSize = atoi(argv[4]);
    // int mapPixelBlockSize = atoi(argv[5]);

    //benchmarking
    FILE *fp; 
    const char *filename = "benchmark.csv";
    fp = fopen(filename, "w");
    fprintf(fp, "histBlockSize, newPixelBlockSize, mapPixelBlockSize, avgHistogramMs, avgNewPixelMs, avgMapPixelMs\n");

    for (int i = 32; i <= 1024; i += 32)
    {
        int histBlockSize = i;
        int newPixelBlockSize = i;
        int mapPixelBlockSize = i;

        float avgHistogramMs = 0;
        float avgNewPixelMs = 0;
        float avgMapPixelMs = 0;

        int runs = 15;
        for (int i = 0; i <= runs; i++){
            //-----------------------------------------
            // Load image from file and allocate space for the output image
            int width, height, cpp;
            unsigned char* h_imageIn = stbi_load(szImage_in_name, &width, &height, &cpp, COLOR_CHANNELS);

            if (h_imageIn == NULL)
            {
                printf("Error reading loading image %s!\n", szImage_in_name);
                exit(EXIT_FAILURE);
            }
            if (cpp != 3)
            {
                printf("Error: Only RGB images supported!\n");
                exit(EXIT_FAILURE);
            }
            // LOG
            // printf("Loaded image %s of size %dx%d.\n", szImage_in_name, width, height);   
            const size_t datasize = width * height * cpp * sizeof(unsigned char);
            unsigned char* h_imageOut = (unsigned char*)malloc(datasize);

            // Calculate size of histogram
            const size_t histogram_size = PIXEL_VALUES * sizeof(int);  

            // Device memory pointers
            unsigned char* d_imageIn;
            unsigned char* d_imageOut;

            int* d_histogram_red;
            int* d_histogram_green;
            int* d_histogram_blue;

            // Allocate memory on the device
            checkCudaErrors(hipMalloc(&d_imageIn, datasize));
            checkCudaErrors(hipMalloc(&d_imageOut, datasize));

            checkCudaErrors(hipMalloc(&d_histogram_red, histogram_size));
            checkCudaErrors(hipMalloc(&d_histogram_green, histogram_size));
            checkCudaErrors(hipMalloc(&d_histogram_blue, histogram_size));

            checkCudaErrors(hipMemset(d_histogram_red, 0, histogram_size));
            checkCudaErrors(hipMemset(d_histogram_green, 0, histogram_size));
            checkCudaErrors(hipMemset(d_histogram_blue, 0, histogram_size));

            // Use CUDA events to measure execution time
            hipEvent_t start, stop, start_overall, stop_overall;
            hipEventCreate(&start);
            hipEventCreate(&stop);
            hipEventCreate(&start_overall);
            hipEventCreate(&stop_overall);

            hipEventRecord(start_overall);

            // Copy input image to device
            // hipEventRecord(start);
            checkCudaErrors(hipMemcpy(d_imageIn, h_imageIn, datasize, hipMemcpyHostToDevice));     

            // float memCpyMs = 0;
            // hipEventRecord(stop);
            // hipEventSynchronize(stop);
            // hipEventElapsedTime(&memCpyMs, start, stop);  

            
            // STEP 1: Compute histogram

            // Setup Thread organization
            int allPixels = width * height * cpp; // Theoretical maximum number of threads
            dim3 blockSizeHist(histBlockSize); // 32 ... 1024 threads per block
            dim3 gridSizeHist((allPixels - 1) / blockSizeHist.x + 1); // can be any size

            hipEventRecord(start);
            calculate_histogram_kernel <<<gridSizeHist, blockSizeHist >>> (d_imageIn, d_histogram_red, d_histogram_green, d_histogram_blue, width, height, cpp);
            hipDeviceSynchronize();

            float histogramMs = 0;
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&histogramMs, start, stop);

        #if DEBUG
            int h_histogram_red[PIXEL_VALUES] = { 0 };
            int h_histogram_green[PIXEL_VALUES] = { 0 };
            int h_histogram_blue[PIXEL_VALUES] = { 0 };

            int h_histogram_red_cpu[PIXEL_VALUES] = { 0 };    
            int h_histogram_green_cpu[PIXEL_VALUES] = { 0 };
            int h_histogram_blue_cpu[PIXEL_VALUES] = { 0 };

            calculate_histogram_cpu(h_imageIn, h_histogram_red_cpu, h_histogram_green_cpu, h_histogram_blue_cpu, width, height, cpp);
            checkCudaErrors(hipMemcpy(h_histogram_red, d_histogram_red, histogram_size, hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy(h_histogram_green, d_histogram_green, histogram_size, hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy(h_histogram_blue, d_histogram_blue, histogram_size, hipMemcpyDeviceToHost));

            for (int i = 0; i < PIXEL_VALUES; i++)
            {
                printf("Values %d R: %d - %d, G: %d - %d, B: %d - %d\n", i, h_histogram_red_cpu[i], h_histogram_red[i], h_histogram_green_cpu[i], h_histogram_green[i], h_histogram_blue_cpu[i], h_histogram_blue[i]);
                assert(h_histogram_red[i] == h_histogram_red_cpu[i]);
                assert(h_histogram_green[i] == h_histogram_green_cpu[i]);
                assert(h_histogram_blue[i] == h_histogram_blue_cpu[i]);
            }
        #endif

            // STEP 2: Compute cumulative sum  
            // calculate_cumulative_sum_simple_kernel <<<gridSize, blockSize >>> (d_histogram_red, d_histogram_green, d_histogram_blue);

            // STEP 2: Optimized cumulative sum computation

            // Setup Thread organization
            dim3 blockSizeCumSum(128); // static
            dim3 gridSizeCumSum(3); // static

            float cumulativeMs = 0;
            hipEventRecord(start);
            calculate_cumulative_sum_kernel <<<gridSizeCumSum, blockSizeCumSum >>> (d_histogram_red, d_histogram_green, d_histogram_blue);
            hipDeviceSynchronize();
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&cumulativeMs, start, stop);

        #if DEBUG
            calculate_cumulative_histogram_cpu(h_histogram_red_cpu, h_histogram_green_cpu, h_histogram_blue_cpu);
            checkCudaErrors(hipMemcpy(h_histogram_red, d_histogram_red, histogram_size, hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy(h_histogram_green, d_histogram_green, histogram_size, hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy(h_histogram_blue, d_histogram_blue, histogram_size, hipMemcpyDeviceToHost));
            for (int i = 0; i < PIXEL_VALUES; i++)
            {
                printf("Values %d R: %d - %d, G: %d - %d, B: %d - %d\n", i, h_histogram_red_cpu[i], h_histogram_red[i], h_histogram_green_cpu[i], h_histogram_green[i], h_histogram_blue_cpu[i], h_histogram_blue[i]);
                assert(h_histogram_red[i] == h_histogram_red_cpu[i]);
                assert(h_histogram_green[i] == h_histogram_green_cpu[i]);
                assert(h_histogram_blue[i] == h_histogram_blue_cpu[i]);
            }
        #endif   

            // STEP 3: Compute new pixel intensities

            // Setup Thread organization
            dim3 blockSizeNewPixel(newPixelBlockSize); // static
            dim3 gridSizeNewPixel((allPixels - 1) / blockSizeNewPixel.x + 1); // static

            float newPixelMs = 0;
            hipEventRecord(start);
            calculate_new_pixel_intensities_kernel <<<gridSizeNewPixel, blockSizeNewPixel >>> (d_histogram_red, d_histogram_green, d_histogram_blue, width, height);
            hipDeviceSynchronize();
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&newPixelMs, start, stop);

        #if DEBUG
            calculate_new_pixel_intensities_cpu(h_histogram_red_cpu, h_histogram_green_cpu, h_histogram_blue_cpu, width, height);
            checkCudaErrors(hipMemcpy(h_histogram_red, d_histogram_red, histogram_size, hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy(h_histogram_green, d_histogram_green, histogram_size, hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy(h_histogram_blue, d_histogram_blue, histogram_size, hipMemcpyDeviceToHost));
            for (int i = 0; i < PIXEL_VALUES; i++)
            {
                printf("Values %d R: %d - %d, G: %d - %d, B: %d - %d\n", i, h_histogram_red_cpu[i], h_histogram_red[i], h_histogram_green_cpu[i], h_histogram_green[i], h_histogram_blue_cpu[i], h_histogram_blue[i]);
                assert(h_histogram_red[i] == h_histogram_red_cpu[i]);
                assert(h_histogram_green[i] == h_histogram_green_cpu[i]);
                assert(h_histogram_blue[i] == h_histogram_blue_cpu[i]);
            }
        #endif
            // STEP 4: Map new pixel intensities to output image

            // Setup Thread organization
            dim3 blockSizeMapPixel(mapPixelBlockSize); 
            dim3 gridSizeMapPixel((allPixels - 1) / blockSizeMapPixel.x + 1); 

            float mapPixelMs = 0;
            hipEventRecord(start);
            hipDeviceSynchronize();
            map_new_pixel_intensities_kernel <<<gridSizeMapPixel, blockSizeMapPixel >>> (d_imageIn, d_imageOut, d_histogram_red, d_histogram_green, d_histogram_blue, width, height, cpp);
            hipDeviceSynchronize();
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&mapPixelMs, start, stop);

            // STEP 5: Copy output image back to host
            // float copyMs = 0;
            // hipEventRecord(start);
            checkCudaErrors(hipMemcpy(h_imageOut, d_imageOut, datasize, hipMemcpyDeviceToHost));
            getLastCudaError("copy_image() execution failed\n");
            // hipEventRecord(stop);
            // hipEventSynchronize(stop);
            // hipEventElapsedTime(&copyMs, start, stop);

            float overallMs = 0;
            hipEventRecord(stop_overall);
            hipEventSynchronize(stop_overall);
            hipEventElapsedTime(&overallMs, start_overall, stop_overall);
            // printf("%0.5f %0.5f %0.5f %0.5f \n", histogramMs, newPixelMs, mapPixelMs, overallMs);

            // Write the output file
            char szImage_out_name_temp[255];
            strncpy(szImage_out_name_temp, szImage_out_name, 255);
            char* token = strtok(szImage_out_name_temp, ".");
            char* FileType = NULL;
            while (token != NULL)
            {
                FileType = token;
                token = strtok(NULL, ".");
            }

            if (!strcmp(FileType, "png"))
                stbi_write_png(szImage_out_name, width, height, cpp, h_imageOut, width * cpp);
            else if (!strcmp(FileType, "jpg"))
                stbi_write_jpg(szImage_out_name, width, height, cpp, h_imageOut, 100);
            else if (!strcmp(FileType, "bmp"))
                stbi_write_bmp(szImage_out_name, width, height, cpp, h_imageOut);
            else
                printf("Error: Unknown image format %s! Only png, bmp, or bmp supported.\n", FileType);

            // Free device memory
            checkCudaErrors(hipFree(d_imageIn));
            checkCudaErrors(hipFree(d_imageOut));
            checkCudaErrors(hipFree(d_histogram_red));
            checkCudaErrors(hipFree(d_histogram_green));
            checkCudaErrors(hipFree(d_histogram_blue));

            // Clean-up events
            hipEventDestroy(start);
            hipEventDestroy(stop);

            // Free host memory
            free(h_imageIn);
            free(h_imageOut);  
            //-----------------------------------------

            avgHistogramMs += histogramMs;
            avgNewPixelMs += newPixelMs;
            avgMapPixelMs += mapPixelMs;
        }
        avgHistogramMs /= runs;
        avgNewPixelMs /= runs;
        avgMapPixelMs /= runs;
        printf("%d, %d, %d, %0.5f, %0.5f, %0.5f\n", histBlockSize, newPixelBlockSize, mapPixelBlockSize, avgHistogramMs, avgNewPixelMs, avgMapPixelMs);
        fflush(stdout);
        fprintf(fp, "%d, %d, %d, %0.5f, %0.5f, %0.5f\n", histBlockSize, newPixelBlockSize, mapPixelBlockSize, avgHistogramMs, avgNewPixelMs, avgMapPixelMs);
    }
    fclose(fp);
    return 0;
}
